#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define INFTY 99999

int **A_Matrix, **D_Matrix;
int **stpSet;

// A utility function to find the vertex with minimum distance value, from 
// the set of vertices not yet included in shortest path tree 
__device__ int minDistance(int* dist, int* stpSet, int n) 
{ 
   // Initialize min value 
    int min = INFTY, min_index; 
    for(int j = 0; j < n; j++){
        if (stpSet[j] == 0 && dist[j] <= min) 
        min = dist[j], min_index = j;
    } 
    return min_index; 
} 


__device__ void makeAdjacency(int n){   //Set initial values to node distances
    int N=n;
    int i,j;
    A_Matrix = (int **)malloc(N * sizeof(int *));
    for (i = 0; i < N; i++)
    {
        A_Matrix[i] = (int *)malloc(N * sizeof(int));
    }
    srand(0);
    for(i = 0; i < N; i++)
    {
        for(j = i; j < N; j++)
        {
            if(i == j){
                A_Matrix[i][j] = 0;
            }
            else{
                int r = rand() % 10;
                int val = (r == 5)? INFTY: r;
                A_Matrix[i][j] = val;
                A_Matrix[j][i] = val; 
            }

        }
    }
}






__global__ void dijkstra_all(int** graph, int** dist, int** stpSet, int n){
    int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    if(tid < n)
        dijkstra(graph, dist[tid], stpSet[tid], n, tid);
}

// Function that implements Dijkstra's single source shortest path algorithm 
// for a graph represented using adjacency matrix representation 
__device__ void dijkstra(int** graph,int* dist,int* stpSet, int n, int src) 
{      // The output array.  dist[i] will hold the shortest 
                      // distance from src to i 
   
    for (int i = 0; i < n; i++) 
        dist[i] = INFTY, stpSet[i] = 0;  
    dist[src] = 0; 
    for (int count = 0; count < n-1; count++) 
    { 
        // Pick the minimum distance vertex from the set of vertices not 
        // yet processed. u is always equal to src in the first iteration. 
        int u = minDistance(dist, stpSet , n); 
        // Mark the picked vertex as processed 
        stpSet[u] = 1; 
        // Update dist value of the adjacent vertices of the picked vertex. 
        for (int v = 0; v < n; v++) 
                 
        // Update dist[v] only if is not in stpSet, there is an edge from  
        // u to v, and total weight of path from src to  v through u is  
        // smaller than current value of dist[v] 
            if (!stpSet[v] && graph[u][v] && dist[u] != INFTY  
                                    && dist[u]+graph[u][v] < dist[v]) 
                dist[v] = dist[u] + graph[u][v]; 
            } 
} 

/* Serial

// driver program to test above function 
int main(int argc, char **argv) 
{ 
    int N;     
	N = atoi(argv[1]);  //Read the console inputs
    int i, j, k;
	D_Matrix =(int **) malloc(N * sizeof(int *));
    stpSet = (int **) malloc(N * sizeof(int *));
    for (i = 0; i < N; i++)
    {
        D_Matrix[i] = (int *)malloc(N * sizeof(int));
        stpSet[i] = (int *)malloc(N * sizeof(int));
    }
    makeAdjacency(N);
    clock_t start = clock();  //First time measurement
    // algorithm -->
    dijkstra_all(A_Matrix,D_Matrix,stpSet,N);
    // <--;	
	clock_t end = clock();   //Final time calculation and convert it into seconds
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Elapsed time = %f sec\n", seconds);
    return 0; 
} 


*/




int main(int argc, char **argv){
    int N;     
	N = atoi(argv[1]);  //Read the console inputs
    int i;
    int *temp;
    temp = (int)malloc(N * sizeof(int));
    for(i = 0; i < N; i++)
    {
        temp[i] = 0; 
    }
	D_Matrix = (int **)malloc(N * sizeof(int *));
    for (i = 0; i < N; i++)
    {
        D_Matrix[i] = (int *)malloc(N * sizeof(int));
    }
    makeAdjacency(N);
	int gridx = pow(2, N - 4), gridy = pow(2, N - 4);  //Dimensions of grid
	int blockx = pow(2, 4), blocky = pow(2, 4);
	dim3 dimGrid(gridx, gridy);
	dim3 dimBlock(blockx, blocky);
	
	// allocate memory on the device
	int* device_dist;
    hipMalloc( (void**)&device_dist, N*N * sizeof (int) );
    hipMalloc( (void**)&device_graph, N*N * sizeof (int));
    hipMalloc( (void**)&device_stpSet, N*N * sizeof (int));
    
    hipMemcpy(device_stpSet, temp, N * sizeof(int), hipMemcpyHostToDevice);
    // initialize dist matrix on device
    for (int i = 0; i < N; i++){
		hipMemcpy(device_dist +i*N, A_Matrix[i], N * sizeof (int),
                              hipMemcpyHostToDevice);
    }
    for (int i = 0; i < N; i++){
		hipMemcpy(device_graph +i*N, A_Matrix[i], N * sizeof (int),
                              hipMemcpyHostToDevice);
    }
    dijkstra_all(int** graph, int** dist, int** stpSet, int n)
	clock_t start = clock();
	dijkstra_all<<<dimGrid,dimBlock>>>(device_graph,device_dist,device_stpSet,N);
	clock_t end = clock();   //Final time calculation and convert it into seconds
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Elapsed time on gpu = %f sec\n", seconds);

	// return results to dist matrix on host
	for (int i = 0; i < N; i++)
		 hipMemcpy(D_Matrix[i], device_dist +i*N, N * sizeof (int),
							  hipMemcpyDeviceToHost);	
}



