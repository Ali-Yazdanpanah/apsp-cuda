#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define MAX_W 9999999
#define TRUE    1
#define FALSE   0
typedef int boolean;
int E,V;
typedef struct
{
	int u;
	int v;
} Edge;

typedef struct 
{
	int title;
	boolean visited;	
} Vertex;

int *weights;
Vertex *vertices;	
Edge *edges;
//Finds the weight of the path from vertex u to vertex v
__device__ __host__ int findEdge(Vertex u, Vertex v, Edge *edges, int *weights, int E)
{
	for(int i = 0; i < E; i++)
	{
		if(edges[i].u == u.title && edges[i].v == v.title)
		{
			return weights[i];
		}
	}
	return MAX_W;
}


__global__ void Find_Vertex(Vertex *vertices, Edge *edges, int *weights, int *length, int *updateLength, int V, int E)
{
	int u = threadIdx.x;
	if(vertices[u].visited == FALSE)
	{
		vertices[u].visited = TRUE;
		for(int v = 0; v < V; v++)
		{				
			int weight = findEdge(vertices[u], vertices[v], edges, weights, E);
			if(weight < MAX_W)
			{	
				if(updateLength[v] > length[u] + weight)
				{
					updateLength[v] = length[u] + weight;
				}
			}
		}

	}
	
}

//Updates the shortest path array (length)
__global__ void Update_Paths(Vertex *vertices, int *length, int *updateLength)
{
	int u = threadIdx.x;
	if(length[u] > updateLength[u])
	{
		vertices[u].visited = FALSE;
		length[u] = updateLength[u];
	}
	updateLength[u] = length[u];
}



void Graph_Randomizer(int V, int E){
	srand(time(NULL));
	
	for(int i = 0; i < V; i++)
	{
		Vertex a = { .title =(int) i, .visited=FALSE};
		vertices[i] = a;
	}
	for(int i = 0; i < E; i++)
	{
		Edge e = {.u = (int) rand()%V , .v = rand()%V};
		edges[i] = e;
		weights[i] = rand()%100;
	}
}

//Runs the program
int main(int argc, char **argv)
{
	V = atoi(argv[1]);
	E = atoi(argv[2]);


	int *len, *updateLength;
	Vertex *d_V;
	Vertex *root;
	Edge *d_E;
	int *d_W;
	int *d_L;
	int *d_C;
	vertices = (Vertex *)malloc(sizeof(Vertex) * V);
    edges = (Edge *)malloc(sizeof(Edge) * E);
    weights = (int *)malloc(E* sizeof(int));
	Graph_Randomizer(V, E);
	len = (int *)malloc(V * sizeof(int));
	updateLength = (int *)malloc(V * sizeof(int));
	root = (Vertex *)malloc(sizeof(Vertex) * V);
	hipMalloc((void**)&d_V, sizeof(Vertex) * V);
	hipMalloc((void**)&d_E, sizeof(Edge) * E);
	hipMalloc((void**)&d_W, E * sizeof(int));
	hipMalloc((void**)&d_L, V * sizeof(int));
    hipMalloc((void**)&d_C, V * sizeof(int));
    hipMemcpy(d_V, vertices, sizeof(Vertex) * V, hipMemcpyHostToDevice);
	hipMemcpy(d_E, edges, sizeof(Edge) * E, hipMemcpyHostToDevice);
	hipMemcpy(d_W, weights, E * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_L, len, V * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, updateLength, V * sizeof(int), hipMemcpyHostToDevice);
	for(int count = 0; count < V; count++){
        root[count].title = count;
        root[count].visited = FALSE;   
    }
    clock_t start = clock();
	for(int count = 0; count < V; count++){
		root[count].visited = TRUE;
		len[root[count].title] = 0;
		updateLength[root[count].title] = 0;
		for(int i = 0; i < V;i++)
		{
			if(vertices[i].title != root[count].title)
			{
                len[(int)vertices[i].title] = findEdge(root[count], vertices[i], edges, weights, E);
				updateLength[vertices[i].title] = len[(int)vertices[i].title];
			}
			else{
				vertices[i].visited = TRUE;
			}
		}	
		hipMemcpy(d_L, len, V * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(d_C, updateLength, V * sizeof(int), hipMemcpyHostToDevice);
		for(int i = 0; i < V; i++){
				Find_Vertex<<<1, V>>>(d_V, d_E, d_W, d_L, d_C, V, E);
				for(int j = 0; j < V; j++)
				{
					Update_Paths<<<1,V>>>(d_V, d_L, d_C);
				}
		}	
    }
	clock_t end = clock();   
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Elapsed time on GPU = %f sec\n", seconds);
}



	
	
	
	
	
	
	
	
	
	
	
	
	

