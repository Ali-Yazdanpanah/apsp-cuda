#include "hip/hip_runtime.h"

#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "floyd_Warshall_serial.h"

#define INFTY 99999

int **A_Matrix, **D_Matrix;




__global__ void floyd_warshall_parallel_kernel(int* dev_dist, int N, int k) {\
	int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	int i, j;
	int dist1, dist2, dist3;
	if (tid < N*N) {
		i = tid / N;
		j = tid - i*N;
		dist1 = dev_dist[tid];
		dist2 = dev_dist[i*N +k];
		dist3 = dev_dist[k*N +j];
		if (dist1 > dist2 + dist3)
			dev_dist[tid] = dist2 + dist3;
	}
}


void makeAdjacency(int n){   //Set initial values to node distances
    int N=n;
    int i,j;
    A_Matrix = malloc(N * sizeof(int *));
    for (i = 0; i < N; i++)
    {
        A_Matrix[i] = malloc(N * sizeof(int));
    }

    srand(0);
    for(i = 0; i < N; i++)
    {
        for(j = i; j < N; j++)
        {
            if(i == j){
                A_Matrix[i][j] = 0;
            }
            else{
                int r = rand() % 10;
                int val = (r == 5)? INFTY: r;
                A_Matrix[i][j] = val;
                A_Matrix[j][i] = val; 
            }

        }
    }
}

int main(int argc, char **argv){
    int N;     
	N = atoi(argv[1]);  //Read the console inputs
    int i, j, k;
	D_Matrix = malloc(N * sizeof(int *));
    for (i = 0; i < N; i++)
    {
        D_Matrix[i] = malloc(N * sizeof(int));
    }
    makeAdjacency(N);
    clock_t start = clock();  //First time measurement
    // algorithm -->
    floyd_warshall_serial(A_Matrix,D_Matrix,N);
    // <--;	
	clock_t end = clock();   //Final time calculation and convert it into seconds
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Elapsed time = %f sec\n", seconds);
	int gridx = pow(2, N - 4), gridy = pow(2, N - 4);  //Dimensions of grid
	int blockx = pow(2, 4), blocky = pow(2, 4);
	dim3 dimGrid(gridx, gridy);
	dim3 dimBlock(blockx, blocky);
	
	
	// allocate memory on the device
	int* device_dist;
	gpuErrchk( hipMalloc( (void**)&device_dist, N*N * sizeof (int) ) );
	// initialize dist matrix on device
	for (int i = 0; i < N; i++)
		gpuErrchk( hipMemcpy(device_dist +i*N, graph[i], N * sizeof (int),
							  hipMemcpyHostToDevice) );
	// For each element of the vertex set
	
	clock_t start = clock();
	for (int k = 0; k < N; k++) {
		// launch kernel
		floyd_warshall_parallel_kernel<<<dimGrid,dimBlock>>>(device_dist,N,k);
		gpuKerchk();
    	}
	clock_t end = clock();   //Final time calculation and convert it into seconds
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Elapsed time on gpu = %f sec\n", seconds);

	// return results to dist matrix on host
	for (int i = 0; i < N; i++)
		 gpuErrchk( hipMemcpy(dist[i], device_dist +i*N, N * sizeof (int),
							  hipMemcpyDeviceToHost) );

	
}



