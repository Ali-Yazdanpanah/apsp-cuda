
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define INFTY 99999

int **A_Matrix, **D_Matrix;




__global__ void floyd_warshall_parallel_kernel(int* dev_dist, int N, int k) {\
	int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	int i, j;
	int dist1, dist2, dist3;
	if (tid < N*N) {
		i = tid / N;
		j = tid - i*N;
		dist1 = dev_dist[tid];
		dist2 = dev_dist[i*N +k];
		dist3 = dev_dist[k*N +j];
		if (dist1 > dist2 + dist3)
			dev_dist[tid] = dist2 + dist3;
	}
}


void makeAdjacency(int n){   //Set initial values to node distances
    int N=n;
    int i,j;
    A_Matrix = (int **)malloc(N * sizeof(int *));
    for (i = 0; i < N; i++)
    {
        A_Matrix[i] = (int *)malloc(N * sizeof(int));
    }
    srand(0);
    for(i = 0; i < N; i++)
    {
        for(j = i; j < N; j++)
        {
            if(i == j){
                A_Matrix[i][j] = 0;
            }
            else{
                int r = rand() % 10;
                int val = (r == 5)? INFTY: r;
                A_Matrix[i][j] = val;
                A_Matrix[j][i] = val; 
            }

        }
    }
}

int main(int argc, char **argv){
    int N;     
	N = atoi(argv[1]);  //Read the console inputs
    int i;
	D_Matrix = (int **)malloc(N * sizeof(int *));
    for (i = 0; i < N; i++)
    {
        D_Matrix[i] = (int *)malloc(N * sizeof(int));
    }
    makeAdjacency(N);
	int gridx = pow(2, N - 4), gridy = pow(2, N - 4);  
	int blockx = pow(2, 4), blocky = pow(2, 4);
	dim3 dimGrid(gridx, gridy);
	dim3 dimBlock(blockx, blocky);
	int* device_dist;
	hipMalloc( (void**)&device_dist, N*N * sizeof (int) );
	for (int i = 0; i < N; i++)
		hipMemcpy(device_dist +i*N, A_Matrix[i], N * sizeof (int),
							  hipMemcpyHostToDevice);
	clock_t start = clock();
	for (int k = 0; k < N; k++) {
		floyd_warshall_parallel_kernel<<<dimGrid,dimBlock>>>(device_dist,N,k);
    	}
	clock_t end = clock();   
	float seconds = (float)(end - start) / CLOCKS_PER_SEC;
	printf("Elapsed time on gpu = %f sec\n", seconds);
	// return results to dist matrix on host
	for (int i = 0; i < N; i++)
		 hipMemcpy(D_Matrix[i], device_dist +i*N, N * sizeof (int),
							  hipMemcpyDeviceToHost);
	
}



