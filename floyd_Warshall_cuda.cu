#include "hip/hip_runtime.h"

/*
 * Max matrix size is N = 2^12 = 4096. 
 * Total number of threads that will be executed is 4096^2.
 * One for each cell.
 */



__global__ void floyd_warshall_parallel_kernel(int* dev_dist, int N, int k) {\
	int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
	int i, j;
	int dist1, dist2, dist3;
	if (tid < N*N) {
		i = tid / N;
		j = tid - i*N;
		dist1 = dev_dist[tid];
		dist2 = dev_dist[i*N +k];
		dist3 = dev_dist[k*N +j];
		if (dist1 > dist2 + dist3)
			dev_dist[tid] = dist2 + dist3;
	}
}
 
// Solves the all-pairs shortest path problem using Floyd Warshall algorithm
void floyd_warshall_parallel(int** graph, int** dist, int N) {

	// allocate memory on the device
	int* device_dist;
	gpuErrchk( hipMalloc( (void**)&device_dist, N*N * sizeof (int) ) );
	// initialize dist matrix on device
	for (int i = 0; i < N; i++)
		gpuErrchk( hipMemcpy(device_dist +i*N, graph[i], N * sizeof (int),
							  hipMemcpyHostToDevice) );
	// For each element of the vertex set
	for (int k = 0; k < N; k++) {
		// launch kernel
		floyd_warshall_parallel_kernel<<<blocks,threads>>>(device_dist,N,k);
		gpuKerchk();
    	}
	// return results to dist matrix on host
	for (int i = 0; i < N; i++)
		 gpuErrchk( hipMemcpy(dist[i], device_dist +i*N, N * sizeof (int),
							  hipMemcpyDeviceToHost) );
}
